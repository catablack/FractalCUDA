#include "hip/hip_runtime.h"
#pragma once
#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/timeb.h>
#include "tinycthread.h"
#include "util.h"
#include <hip/hip_runtime.h>
#include <>

/* This should be conveted into a GPU kernel */
__global__ void generate_image(unsigned char* dev_image, unsigned char* dev_colormap)
{
	int row, col, index, iteration;
	double c_re, c_im, x, y, x_new;

	unsigned char* image = dev_image;
	unsigned char* colormap = dev_colormap;

	int width = WIDTH;
	int height = HEIGHT;
	int max = MAX_ITERATION;

	int blockId = blockIdx.y * gridDim.x + blockIdx.x; // global block id
	index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;// pixel index for the thread 

	if (index >= width * height) return;

	row = index / WIDTH;
	col = index % WIDTH;

	c_re = (col - width / 2.0) * 4.0 / width;
	c_im = (row - height / 2.0) * 4.0 / width;

	x = 0, y = 0;
	iteration = 0;
	while (x * x + y * y <= 4 && iteration < max) {
		x_new = x * x - y * y + c_re;
		y = 2 * x * y + c_im;
		x = x_new;
		iteration++;
	}

	if (iteration > max) {
		iteration = max;
	}

	image[4 * index + 0] = colormap[iteration * 3 + 0];
	image[4 * index + 1] = colormap[iteration * 3 + 1];
	image[4 * index + 2] = colormap[iteration * 3 + 2];
	image[4 * index + 3] = 255;

}

int main(int argc, char** argv) {
	
	double times[REPEAT];
	struct timeb start[REPEAT], end[REPEAT];
	char path[255];

	unsigned char* colormap;
	unsigned char* image;

	hipMallocManaged(&colormap, (MAX_ITERATION + 1) * 3);
	hipMallocManaged(&image, WIDTH * HEIGHT * 4);

	init_colormap(MAX_ITERATION, colormap);

	dim3 grid(GRID_SIZE_X, GRID_SIZE_Y);
	dim3 block(BLOCK_SIZE_X, BLOCK_SIZE_Y);

	for (int i = 0; i < REPEAT; i++) {
		ftime(&start[i]);
		
		generate_image <<<grid, block >>> (image, colormap);

		hipDeviceSynchronize();
		
		ftime(&end[i]);
		
		times[i] =  end[i].time - start[i].time + ((double)end[i].millitm - (double)start[i].millitm) / 1000.0;

		sprintf(path, IMAGE, "gpu", i);
		save_image(path, image, WIDTH, HEIGHT);
		progress("gpu", i, times[i]);

	}

	report("gpu", times);
	
	printf("\nDONE!!!");

	hipFree(image);
	hipFree(colormap);

	return 0;
}
